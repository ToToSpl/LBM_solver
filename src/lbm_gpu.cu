#include "hip/hip_runtime.h"

#include <cstddef>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/types.h>

#include "../include/lbm_gpu.cuh"
#include "./lbm_helpers.cuh"

// ---- KERNELS -------

__device__ inline u_int32_t get_index(LatticeInfo &space_data, u_int32_t x,
                                      u_int32_t y, u_int32_t z) {
  return (z * space_data.x_size * space_data.y_size) + (y * space_data.x_size) +
         x;
}

__global__ void gpu_init_memory(LatticeNode *space, LatticeInfo space_data) {

  u_int32_t x = blockDim.x * blockIdx.x + threadIdx.x;
  u_int32_t y = blockDim.y * blockIdx.y + threadIdx.y;
  u_int32_t z = blockDim.z * blockIdx.z + threadIdx.z;

  if (!(x < space_data.x_size && y < space_data.y_size &&
        z < space_data.z_size))
    return;

  u_int32_t index = get_index(space_data, x, y, z);

  space[index].f[0] = index;
}

// ---- END KERNELS -------

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

void cuda_wait_for_device() { gpuErrchk(hipDeviceSynchronize()); }

void lbm_space_init_device(LatticeSpace *space) {
  gpuErrchk(hipMalloc(&space->device_data,
                       space->info.total_size * sizeof(LatticeNode)));
}

void lbm_space_init_kernel(LatticeSpace *space) {
  ComputeDim compute_dim = compute_dim_create(
      space->info.x_size, space->info.y_size, space->info.z_size);

  gpu_init_memory<<<compute_dim.gridSize, compute_dim.blockSize>>>(
      space->device_data, space->info);
  gpuErrchk(hipPeekAtLastError());
}

LatticeNode *lbm_space_copy_host(LatticeSpace *space) {
  LatticeNode *raw_data =
      (LatticeNode *)malloc(sizeof(LatticeNode) * space->info.total_size);
  gpuErrchk(hipMemcpy(raw_data, space->device_data,
                       space->info.total_size * sizeof(LatticeNode),
                       hipMemcpyDeviceToHost));
  return raw_data;
}
