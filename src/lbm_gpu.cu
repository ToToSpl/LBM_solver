#include "hip/hip_runtime.h"

#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../include/lbm_gpu.cuh"

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void gpu_init_memory(LatticeNode *space, LatticeInfo space_data) {
  u_int32_t x = blockDim.x * blockIdx.x + threadIdx.x;
  u_int32_t y = blockDim.y * blockIdx.y + threadIdx.y;
  u_int32_t z = blockDim.z * blockIdx.z + threadIdx.z;

  if (x < space_data.x_size && y < space_data.y_size && z < space_data.z_size) {
    u_int32_t index = (z * space_data.x_size * space_data.y_size) +
                      (y * space_data.x_size) + x;
    float pos = (float)index;
    space[index].f[0] = 69.f;
    /*{pos, pos, pos, pos, pos, pos, pos, pos, pos,
                  pos, pos, pos, pos, pos, pos, pos, pos, pos,
                  pos, pos, pos, pos, pos, pos, pos, pos, pos};*/
  }
}

void cuda_wait_for_device() { gpuErrchk(hipDeviceSynchronize()); }

void lbm_space_init_device(LatticeSpace *space) {
  space->device_data = malloc(sizeof(hipPitchedPtr));
  hipExtent volumeSizeBytes =
      make_hipExtent(sizeof(LatticeNode) * space->info.x_size,
                      space->info.y_size, space->info.z_size);
  gpuErrchk(
      hipMalloc3D((hipPitchedPtr *)space->device_data, volumeSizeBytes));
}

void lbm_space_init_kernel(LatticeSpace *space) {
  dim3 blockSize(3, 3, 3);
  dim3 gridSize(1, 1, 1);
  gpu_init_memory<<<gridSize, blockSize>>>(
      (LatticeNode *)((hipPitchedPtr *)space->device_data)->ptr, space->info);
  gpuErrchk(hipPeekAtLastError());
}

void lbm_space_copy_host(LatticeNode *raw_data, LatticeSpace *space) {
  raw_data =
      (LatticeNode *)malloc(sizeof(LatticeNode) * space->info.total_size);
  gpuErrchk(hipMemcpy(raw_data, ((hipPitchedPtr *)space->device_data)->ptr,
                       sizeof(LatticeNode) * space->info.total_size,
                       hipMemcpyDeviceToHost));
}
